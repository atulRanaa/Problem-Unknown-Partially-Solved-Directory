
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void helloFromGPU(){
	printf("Hello World from GPU: %d\n",threadIdx.x);
}
int main(void){
	helloFromGPU<<<1,10>>>();
	hipDeviceReset();
	//cudaDeviceSynchronize();
	return 0;
}
