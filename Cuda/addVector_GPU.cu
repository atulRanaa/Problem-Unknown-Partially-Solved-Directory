
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

int N = 1<<10;
__global__ void add(int *a,int *b,int n){
	int index = threadIdx.x;
	if( index < n)
		b[index] = a[index]+b[index];
}

int main(void){
	
	int *A,*B;
	int *a,*b;
	
	A = (int*)malloc(N*sizeof(int));
	B = (int*)malloc(N*sizeof(int));

	hipMalloc(&a, N*sizeof(int));
	hipMalloc(&b, N*sizeof(int));
	
	for(int i=0;i<N;i++){
		A[i]=-i;
		B[i]=i*i;
	}
	hipMemcpy(a, A, N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(b, B, N*sizeof(int), hipMemcpyHostToDevice);
	
	add<<<1,512>>>(a,b,N);
	
	hipMemcpy(B, b, N*sizeof(int), hipMemcpyDeviceToHost); 

	hipFree(a);
	hipFree(b);

	free(A);
	free(B);

	return 0;
}
