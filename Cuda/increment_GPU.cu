
#include <hip/hip_runtime.h>
#include<stdio.h>
//#include"gputimer.h"

#define NUM_THREADS 100000
#define ARRAY_SIZE 10
#define BLOCK_WIDTH 1000

__global__ void increment_naive(int *g){
	int i = blockIdx.x*blockDim.x + threadIdx.x;

	i = i%ARRAY_SIZE;
	g[i] = g[i]+1;
}
__global__ void increment_atomic(int *g){
	int i = blockIdx.x*blockDim.x + threadIdx.x;

	i = i%ARRAY_SIZE;
	atomicAdd(&g[i], 1);
}
int main(){

	//GpuTimer timer;
	printf("%d total threads in %d blocks writing into %d array elements\n",
	NUM_THREADS,NUM_THREADS/BLOCK_WIDTH,ARRAY_SIZE);

	int h_array[ARRAY_SIZE];
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(int);

	int *d_array;
	hipMalloc((void **)&d_array,ARRAY_BYTES);
	hipMemset((void **)d_array,0,ARRAY_BYTES);

	//timer.start();
	//increment_naive<<< NUM_THREADS/BLOCK_WIDTH,BLOCK_WIDTH >>>(d_array);
	increment_atomic<<< NUM_THREADS/BLOCK_WIDTH,BLOCK_WIDTH >>>(d_array);
	//timer.stop();

	hipMemcpy(h_array,d_array,ARRAY_BYTES,hipMemcpyDeviceToHost);
	//printf("Time elapsed = %g ms\n",timer.Elapsed());
	for(int i=0;i< ARRAY_SIZE ; i++)	printf("%d ", h_array[i]);
	hipFree(d_array);
	return 0;
}
